// -*- c++ -*-


#include <iostream>

#include "qdp_config_internal.h" 

#include "qdp_cuda.h"
#include "qdp_init.h"

using namespace std;



namespace QDP {


  void inline cudp_check_error(const std::string& msg,hipError_t& ret)
  {
#ifdef GPU_DEBUG_DEEP
    QDP_info("%s\n", msg.c_str());

    if (ret != hipSuccess) {
      string tmp(hipGetErrorString(ret));
      QDP_error_exit("qdp_cuda.cu: %s",tmp.c_str());
    }

    //CudaThreadSynchronize();
    hipError_t error = hipGetLastError();
    if (ret != hipSuccess) {
      string tmp(hipGetErrorString(ret));
      QDP_error_exit("qdp_cuda.cu: %s",tmp.c_str());
    }

#else
    if (ret != hipSuccess) {
      QDP_info("%s\n",msg.c_str());
      string tmp(hipGetErrorString(ret));
      QDP_error_exit("qdp_cuda.cu: %s",tmp.c_str());
    }
#endif
  }


  hipStream_t * QDPcudastreams;
  hipEvent_t * QDPevCopied;

  void * CudaGetKernelStream() {
    return (void*)&QDPcudastreams[KERNEL];
  }

  void CudaCreateStreams() {
    QDPcudastreams = new hipStream_t[2];
    for (int i=0; i<2; i++) {
      QDP_info("Creating CUDA stream %d",i);
      hipStreamCreate(&QDPcudastreams[i]);
    }
    QDP_info("Creating CUDA event for transfers");
    QDPevCopied = new hipEvent_t;
    hipEventCreate(QDPevCopied);

  }

  void CudaSyncKernelStream() {
    hipStreamSynchronize(QDPcudastreams[KERNEL]);
  }

  void CudaSyncTransferStream() {
    hipStreamSynchronize(QDPcudastreams[TRANSFER]);
  }

  void CudaRecordAndWaitEvent() {
    hipEventRecord( *QDPevCopied , QDPcudastreams[TRANSFER] );
    hipStreamWaitEvent( QDPcudastreams[KERNEL] , *QDPevCopied , 0);
  }

  void CudaSetDevice(int dev)
  {
    hipError_t ret;
    ret = hipSetDevice(dev);
    cudp_check_error("hipSetDevice",ret);
  }

  void CudaGetDeviceCount(int * count)
  {
    hipError_t ret;
    ret = hipGetDeviceCount( count );
    cudp_check_error("hipGetDeviceCount",ret);
  }


  bool CudaHostRegister(void * ptr , size_t size)
  {
    hipError_t ret;
    //int flags = hipHostMallocWriteCombined | hipHostRegisterPortable;
    int flags = 0;
    QDP_info("CUDA host register ptr=%p (%u) size=%lu (%u)",ptr,(unsigned)((size_t)ptr%4096) ,(unsigned long)size,(unsigned)((size_t)size%4096));
    ret = hipHostRegister(ptr, size, flags);
    cudp_check_error("hostRegister",ret);
    return true;
  }

  
  void CudaHostUnregister(void * ptr )
  {
    hipError_t ret;
    ret = hipHostUnregister(ptr);
    cudp_check_error("hostUnregister",ret);
  }
  
  void CudaMemGetInfo(size_t *free,size_t *total)
  {
    hipError_t ret;
    ret = hipMemGetInfo( free , total );
    cudp_check_error("getMemInfo",ret);
  }



  bool CudaHostAlloc(void **mem , const size_t size, const int flags)
  {
    hipError_t ret;
    ret = hipHostAlloc(mem,size,flags);
    cudp_check_error("hipHostAlloc",ret);
    return ret == hipSuccess;
  }


  void CudaHostAllocWrite(void **mem , size_t size)
  {
    hipError_t ret;
    ret = hipHostAlloc(mem,size,hipHostMallocWriteCombined);
    cudp_check_error("hipHostAlloc write_combined",ret);
  }


  void CudaHostFree(const void *mem)
  {
    hipError_t ret;
    ret = hipHostFree((void *)mem);
    cudp_check_error("hipHostFree",ret);
  }




  void CudaMemcpy( const void * dest , const void * src , size_t size)
  {
    hipError_t ret;
#ifdef GPU_DEBUG_DEEP
    QDP_debug_deep("hipMemcpy dest=%p src=%p size=%d" ,  dest , src , size );
#endif
    //QDP_info("hipMemcpy dest=%p src=%p size=%d" ,  dest , src , size );

    ret = hipMemcpy(const_cast<void*>(dest),
		     const_cast<void*>(src),
		     size,hipMemcpyDefault);

    cudp_check_error("hipMemcpy",ret);
  }


  void CudaMemcpyAsync( const void * dest , const void * src , size_t size )
  {
    hipError_t ret;
#ifdef GPU_DEBUG_DEEP
    QDP_debug_deep("hipMemcpy dest=%p src=%p size=%d" ,  dest , src , size );
#endif

    ret = hipMemcpyAsync(const_cast<void*>(dest),
			  const_cast<void*>(src),
			  size,hipMemcpyDefault,
			  QDPcudastreams[TRANSFER]);

    cudp_check_error("hipMemcpy",ret);
  }


  bool CudaMalloc(void **mem , size_t size )
  {
    hipError_t ret;
    ret = hipMalloc(mem,size);
#ifdef GPU_DEBUG_DEEP
    QDP_debug_deep( "CudaMalloc %p", *mem );
#endif
    return ret == hipSuccess;
  }

  void CudaFree(const void *mem )
  {
#ifdef GPU_DEBUG_DEEP
    QDP_debug_deep( "CudaFree %p", mem );
#endif
    hipError_t ret;
    ret = hipFree(const_cast<void*>(mem));
    cudp_check_error("hipFree",ret);
  }

  void CudaThreadSynchronize()
  {
#ifdef GPU_DEBUG_DEEP
    QDP_debug_deep( "hipDeviceSynchronize" );
#endif
    hipDeviceSynchronize();
  }

  void CudaDeviceSynchronize()
  {
#ifdef GPU_DEBUG_DEEP
    QDP_debug_deep( "hipDeviceSynchronize" );
#endif
    hipDeviceSynchronize();
  }

}


